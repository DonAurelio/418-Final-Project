#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <thrust/scan.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "34-2.h"
#include "util.h"

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct global_constants {
  int grid_width;
  int grid_height;
  grid_elem* curr_grid;
  grid_elem* next_grid;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ global_constants const_params;

// kernelClearGrid --  (CUDA device code)
//
// Clear the grid, setting all cells to 0
__global__ void kernel_clear_grid() {

  // cells at border are not modified
  int image_x = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int image_y = blockIdx.y * blockDim.y + threadIdx.y + 1;

  int width = const_params.grid_width;
  int height = const_params.grid_width;

  // cells at border are not modified
  if (image_x >= width - 1 || image_y >= height - 1)
      return;

  int offset = image_y*width + image_x;

  // write to global memory
  const_params.curr_grid[offset] = 0;
}

#define THREAD_DIMX 32
#define THREAD_DIMY 16


// kernel_single_iteration (CUDA device code)
//
// compute a single iteration on the grid, putting the results in next_grid
__global__ void kernel_single_iteration(grid_elem* curr_grid, grid_elem* next_grid) {

  // all the cells which need to be looked at by this block
  __shared__ float local_cells[THREAD_DIMX*THREAD_DIMY];

  // remember that cells on the border of a block don't do any work
  int image_x = blockIdx.x * (THREAD_DIMX - 2) + threadIdx.x;
  int image_y = blockIdx.y * (THREAD_DIMY - 2) + threadIdx.y;

  int width = const_params.grid_width;
  int height = const_params.grid_width;

  // index in the grid of this thread
  int grid_index = image_y*width + image_x;

  // index in the grid of local cells
  int local_index = threadIdx.y*THREAD_DIMX + threadIdx.x;

  // store data in shared memory

  // only threads inside of the range of the grid should store shared memory
  if (image_x < width && image_y < height) {
    local_cells[local_index] = curr_grid[grid_index];
  }
  __syncthreads();

  // the algorithm only computes the next state of cells:
  //    -in a valid grid cell, but not in the border of the grid
  //    -not in the border of any blocks
  if (image_x < width - 1 && image_y < height - 1 &&
      0 < threadIdx.x && threadIdx.x < THREAD_DIMX - 1 &&
      0 < threadIdx.y && threadIdx.y < THREAD_DIMY - 1) {


    uint8_t live_neighbors = 0;

    grid_elem curr_value = local_cells[local_index];

    // compute the number of live_neighbors

    //{up, up-right, left, right, down-left, down}

    live_neighbors += local_cells[local_index - THREAD_DIMX];
    live_neighbors += local_cells[local_index - THREAD_DIMX + 1];
    live_neighbors += local_cells[local_index - 1];
    live_neighbors += local_cells[local_index + 1];
    live_neighbors += local_cells[local_index + THREAD_DIMX - 1];
    live_neighbors += local_cells[local_index + THREAD_DIMX];

    // values for the next iteration
    grid_elem next_value;

    if (!curr_value) {
      next_value = (live_neighbors == 2);
    } else {
      next_value = (live_neighbors == 3 || live_neighbors == 4);
    }

    //const_params.next_grid[grid_index] = next_value;
    next_grid[grid_index] = next_value;
  }

}


Automaton34_2::Automaton34_2() {
  num_iters = 0;
  grid = NULL;
  cuda_device_grid_curr = NULL;
  cuda_device_grid_next = NULL;
}

Automaton34_2::~Automaton34_2() {
  if (grid) {
    delete grid->data;
    delete grid;
  }
  if (cuda_device_grid_curr) {
    hipFree(cuda_device_grid_curr);
    hipFree(cuda_device_grid_next);
  }
}

Grid*
Automaton34_2::get_grid() {

  // need to copy contents of the final grid from device memory
  // before we expose it to the caller

  printf("Copying grid data from device\n");

  hipMemcpy(grid->data,
             cuda_device_grid_curr,
             sizeof(grid_elem) * grid->width * grid->height,
             hipMemcpyDeviceToHost);

  return grid;
}

void
Automaton34_2::setup(int num_of_iters) {

  int deviceCount = 0;
  bool isFastGPU = false;
  std::string name;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("Number of iterations: %d\n", num_of_iters);
  num_iters = num_of_iters;

  printf("---------------------------------------------------------\n");
  printf("Initializing CUDA for CudaRenderer\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i=0; i<deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    name = deviceProps.name;
    if (name.compare("GeForce GTX 480") == 0
        || name.compare("GeForce GTX 670") == 0
        || name.compare("GeForce GTX 780") == 0)
    {
      isFastGPU = true;
    }

    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   Memory Clock Rate (KHz): %d\n", deviceProps.memoryClockRate);
    printf("   Memory Bus Width (bits): %d\n", deviceProps.memoryBusWidth);
    printf("   Peak Memory Bandwidth (GB/s): %f\n",
        2.0*deviceProps.memoryClockRate*(deviceProps.memoryBusWidth/8)/1.0e6);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);

    printf("---------------------------------------------------------\n");
    if (!isFastGPU) {

      printf("WARNING: "
             "You're not running on a fast GPU, please consider using "
             "NVIDIA GTX 480, 670 or 780.\n");
      printf("---------------------------------------------------------\n");
    }
  }

  // By this time the scene should be loaded.  Now copy all the key
  // data structures into device memory so they are accessible to
  // CUDA kernels

  hipMalloc(&cuda_device_grid_curr, sizeof(grid_elem) * grid->width * grid->height);
  hipMalloc(&cuda_device_grid_next, sizeof(grid_elem) * grid->width * grid->height);

  hipMemcpy(cuda_device_grid_curr, grid->data,
              sizeof(grid_elem) * grid->width * grid->height, hipMemcpyHostToDevice);
  hipMemset(cuda_device_grid_next, 0, sizeof(grid_elem) * grid->width * grid->height);

  // Initialize parameters in constant memory.
  global_constants params;
  params.grid_height = grid->height;
  params.grid_width = grid->width;
  params.curr_grid = cuda_device_grid_curr;
  params.next_grid = cuda_device_grid_next;

  hipMemcpyToSymbol(HIP_SYMBOL(const_params), &params, sizeof(global_constants));
}


// create the initial grid using the input file
//
// pattern_x and pattern_y determine how many times the input grid is repeated in the
// x and y directions
void
Automaton34_2::create_grid(char *filename, int pattern_x, int pattern_y) {

  FILE *input = NULL;
  int width, height; // width and height of entire image
  int section_width, section_height; // width and height of the input grid
  grid_elem *data;

  input = fopen(filename, "r");
  if (!input) {
    printf("Unable to open file: %s\n", filename);
    printf("\nTerminating program\n");
    exit(1);
  }

  // copy in width and height from file
  if (fscanf(input, "%d %d\n", &section_width, &section_height) != 2) {
    fclose(input);
    printf("Invalid input\n");
    printf("\nTerminating program\n");
    exit(1);
  }

  width = section_width*pattern_x;
  height = section_height*pattern_y;

  printf("Width: %d\nHeight: %d\n", width, height);

  // increase grid size to account for border cells
  width += 2;
  height += 2;
  data = new grid_elem [width*height];

  // insert data from file into grid
  for (int section_y = 0; section_y < section_height; section_y++) {
    for (int section_x = 0; section_x < section_width; section_x++) {

      int temp;
      if (fscanf(input, "%d", &temp) != 1) {
        fclose(input);
        printf("Invalid input\n");
        printf("\nTerminating program\n");
        exit(1);
      }

      // write value for each pattern
      for (int py = 0; py < pattern_y; py++) {
        for (int px = 0; px < pattern_x; px++) {
          int y_index = py*section_height + section_y + 1;
          int x_index = px*section_width + section_x + 1;
          data[y_index*width + x_index] = (grid_elem)temp;
        }
      }
    }
  }

  fclose(input);

  grid = new Grid(width, height);
  grid->data = data;
}

void
Automaton34_2::run_automaton() {

  // number of threads needed in the x and y directions
  // note that this is less than the width/height due to the border of unmodified cells
  int width_cells = grid->width - 2;
  int height_cells = grid->height - 2;

  // block/grid size for the pixel kernal
  dim3 cell_block_dim(THREAD_DIMX, THREAD_DIMY);
  dim3 cell_grid_dim((width_cells + (cell_block_dim.x - 2) - 1) / (cell_block_dim.x - 2),
              (height_cells + (cell_block_dim.y - 2) - 1) / (cell_block_dim.y - 2));

  for (int iter = 0; iter < num_iters; iter++) {
    kernel_single_iteration<<<cell_grid_dim, cell_block_dim>>>( cuda_device_grid_curr, cuda_device_grid_next);
    hipDeviceSynchronize();
    //hipMemcpy(cuda_device_grid_curr, cuda_device_grid_next,
      //sizeof(grid_elem) * grid->width * grid->height, hipMemcpyDeviceToDevice);
    grid_elem* temp = cuda_device_grid_curr;
    cuda_device_grid_curr = cuda_device_grid_next;
    cuda_device_grid_next = temp;

  }
}
