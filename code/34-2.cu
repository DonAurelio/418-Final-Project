#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <thrust/scan.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "grid.h"
#include "util.h"

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct global_constants {
  int grid_width;
  int grid_height;
  grid_elem* grid;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ global_constants const_params;

// kernelClearGrid --  (CUDA device code)
//
// Clear the grid, setting all cells to 0
__global__ void kernel_clear_grid() {

  // cells at border are not modified
  int image_x = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int image_y = blockIdx.y * blockDim.y + threadIdx.y + 1;

  int width = const_params.grid_width;
  int height = const_params.grid_width;

  // cells at border are not modified
  if (image_x >= width - 1 || image_y >= height - 1)
      return;

  int offset = image_y*width + image_x;

  // write to global memory
  *(grid_elem*)(&global_constants.grid[offset]) = DEAD;
}

#define THREAD_DIMX 32
#define THREAD_DIMY 32


// kernelSingleCycle -- (CUDA device code)
//
// fill in each array of blockCircles so that it represents which circles
// overlap with each block
__global__ void kernalComputeLocalCircles(int *blockCircles, int arraySize) {

    // index of this block in blockCircles
    int blockIndex = blockIdx.y*gridDim.x + blockIdx.x;
    // circle number of the circle this thread will deal with
    int circleIndex = blockIdx.z*blockDim.x + threadIdx.x;

    int numCircles = cuConstRendererParams.numCircles;

    // only continue if circleIndex is valid
    if (circleIndex < numCircles) {

        float3 p = *(float3*) (&cuConstRendererParams.position[circleIndex*3]);
        float  rad = cuConstRendererParams.radius[circleIndex];
        short imageWidth = cuConstRendererParams.imageWidth;
        short imageHeight = cuConstRendererParams.imageHeight;

        int minPixelX = THREAD_DIMX*blockIdx.x;
        int maxPixelX = THREAD_DIMX*(blockIdx.x + 1);
        int minPixelY = THREAD_DIMY*blockIdx.y;
        int maxPixelY = THREAD_DIMY*(blockIdx.y + 1);

        // compute wether circle can intersect with the current block
        int inRange = 1;
        inRange = inRange &&
            maxPixelY > static_cast<short>(imageHeight * (p.y - rad));
        inRange = inRange &&
            minPixelY <=  static_cast<short>(imageHeight * (p.y + rad));
        inRange = inRange &&
            maxPixelX > static_cast<short>(imageWidth * (p.x - rad));
        inRange = inRange &&
            minPixelX <=  static_cast<short>(imageWidth * (p.x + rad));

        // in order to signal the last position in the array, the last
        // circle being rendered is put into every array in blockCircles
        inRange = inRange || (circleIndex == numCircles - 1);
        blockCircles[blockIndex*arraySize + circleIndex] = inRange;
    }


}

// create the keys vector
__global__ void
kernalComputeKeys(int *keys, int arraySize) {

    // get the keys array specific to this block
    keys += blockIdx.y*arraySize;
    // index of this thread in the keys array
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if (index < arraySize) {
        keys[index] = blockIdx.y;
    }
}

__global__ void
kernalCreateCircleList(int *blockCircles, int *blockCirclesScan,
                        int *circleList, int arraySize) {
    // get the arrays specific to this block
    blockCircles += blockIdx.y*arraySize;
    blockCirclesScan += blockIdx.y*arraySize;
    circleList += blockIdx.y*arraySize;

    // index of this thread in any of the arrays
    int listIndex = blockIdx.x*blockDim.x + threadIdx.x;

    int numCircles = cuConstRendererParams.numCircles;

    if (listIndex < numCircles) {
        if (blockCircles[listIndex]) {
            circleList[blockCirclesScan[listIndex]] = listIndex;
        }

    }
}

////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce GTX 480") == 0
            || name.compare("GeForce GTX 670") == 0
            || name.compare("GeForce GTX 780") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA GTX 480, 670 or 780.\n");
        printf("---------------------------------------------------------\n");
    }

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) {
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}

void
CudaRenderer::render() {

    // block/grid size for the pixel kernal
    dim3 pixelBlockDim(THREAD_DIMX, THREAD_DIMY);
    dim3 pixelGridDim((image->width + pixelBlockDim.x - 1) / pixelBlockDim.x,
                (image->height + pixelBlockDim.y - 1) / pixelBlockDim.y);

    // size of each circle array
    int arraySize = numCircles;

    // skip parts of the implementation if there aren't many circles
    int quick = (numCircles < 30);

    // initialize blockCircles so it has enough space to hold each array
    int numBlocks = pixelGridDim.x*pixelGridDim.y;

    if (!quick) {
        // contains the arrays of circles which can overlap with each block
        thrust::device_ptr<int> blockCircles =
            thrust::device_malloc<int>(numBlocks*arraySize);
        // will contain the exclusive scan of blockCircles
        thrust::device_ptr<int> blockCirclesScan =
            thrust::device_malloc<int>(numBlocks*arraySize);
        // contains keys for the exclusive scan
        thrust::device_ptr<int> keys =
            thrust::device_malloc<int>(numBlocks*arraySize);
        // will contain arrays of all circles that overlap with each block
        thrust::device_ptr<int> circleList =
            thrust::device_malloc<int>(numBlocks*arraySize);

        // block/gird size for setting up blockCircles
        dim3 circleBlockDim(512);
        dim3 circleGridDim(pixelGridDim.x, pixelGridDim.y,
                (numCircles + circleBlockDim.x - 1) / circleBlockDim.x);

        // fill in each array of blockCircles so that it represents which
        // circles overlap with each block
        kernalComputeLocalCircles<<<circleGridDim, circleBlockDim>>>
                (blockCircles.get(), arraySize);
        hipDeviceSynchronize();

        // block/gird size for setting up keys
        dim3 keyBlockDim(512);
        dim3 keyGridDim((arraySize + keyBlockDim.x - 1)/keyBlockDim.x,
                numBlocks);

        // create the keys vector
        kernalComputeKeys<<<keyGridDim, keyBlockDim>>>(keys.get(), arraySize);

        thrust::exclusive_scan_by_key(keys, keys+arraySize*numBlocks,
                blockCircles, blockCirclesScan);

        // block/gird size for circleList
        dim3 listBlockDim(512);
        dim3 listGridDim((numCircles + listBlockDim.x - 1) / listBlockDim.x,
                numBlocks);

        // fill in circeList
        kernalCreateCircleList<<<listGridDim, listBlockDim>>>
            (blockCircles.get(), blockCirclesScan.get(),
             circleList.get(), arraySize);
        hipDeviceSynchronize();

        // shade the pixels based on which circles overlap with them
        kernelRenderPixels<<<pixelGridDim, pixelBlockDim>>>
            (circleList.get(), arraySize);

        hipDeviceSynchronize();
        thrust::device_free(circleList);
        thrust::device_free(keys);
        thrust::device_free(blockCirclesScan);
        thrust::device_free(blockCircles);
    } else {
        kernelRenderSmall<<<pixelGridDim, pixelBlockDim>>>();
        hipDeviceSynchronize();
    }
}
